// kernels/cuda/hello.cu
#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void vector_add(float *a, float *b, float *c, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
        c[i] = a[i] + b[i];
}

int main()
{
    printf("Hello from CUDA!\n");

    // Simple CUDA runtime check with detailed error reporting
    hipError_t err = hipSetDevice(0);
    if (err != hipSuccess)
    {
        printf("CUDA error in cudaSetDevice: %s (code %d)\n", hipGetErrorString(err), err);
        printf("This might be due to:\n");
        printf("1. No CUDA-capable GPU found\n");
        printf("2. CUDA driver/runtime version mismatch\n");
        printf("3. GPU in use by another process\n");
        printf("4. Insufficient permissions\n");
        return 1;
    }

    printf("CUDA device 0 set successfully!\n");

    // Get device properties
    hipDeviceProp_t prop;
    err = hipGetDeviceProperties(&prop, 0);
    if (err != hipSuccess)
    {
        printf("Error getting device properties: %s\n", hipGetErrorString(err));
        return 1;
    }

    printf("GPU: %s\n", prop.name);
    printf("Compute capability: %d.%d\n", prop.major, prop.minor);

    // If we get here, try a simple vector addition
    const int N = 8;
    const size_t bytes = N * sizeof(float);

    // Host arrays
    float h_a[N], h_b[N], h_c[N];

    // Initialize input arrays
    for (int i = 0; i < N; i++)
    {
        h_a[i] = i * 1.0f;
        h_b[i] = i * 2.0f;
        h_c[i] = 0.0f; // Initialize output
    }

    printf("Attempting GPU computation...\n");

    // Device arrays
    float *d_a = NULL, *d_b = NULL, *d_c = NULL;

    // Variables declared here to avoid goto scope issues
    int blockSize = 32;
    int gridSize = (N + blockSize - 1) / blockSize;

    // Try to allocate device memory
    err = hipMalloc(&d_a, bytes);
    if (err != hipSuccess)
    {
        printf("hipMalloc failed for d_a: %s\n", hipGetErrorString(err));
        return 1;
    }

    err = hipMalloc(&d_b, bytes);
    if (err != hipSuccess)
    {
        printf("hipMalloc failed for d_b: %s\n", hipGetErrorString(err));
        hipFree(d_a);
        return 1;
    }

    err = hipMalloc(&d_c, bytes);
    if (err != hipSuccess)
    {
        printf("hipMalloc failed for d_c: %s\n", hipGetErrorString(err));
        hipFree(d_a);
        hipFree(d_b);
        return 1;
    }

    printf("GPU memory allocated successfully!\n");

    // Copy data to device
    err = hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("hipMemcpy failed for d_a: %s\n", hipGetErrorString(err));
        goto cleanup;
    }

    err = hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("hipMemcpy failed for d_b: %s\n", hipGetErrorString(err));
        goto cleanup;
    }

    printf("Data copied to GPU successfully!\n");

    // Launch kernel
    printf("Launching kernel with grid=%d, block=%d\n", gridSize, blockSize);

    vector_add<<<gridSize, blockSize>>>(d_a, d_b, d_c, N);

    err = hipPeekAtLastError();
    if (err != hipSuccess)
    {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
        goto cleanup;
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess)
    {
        printf("hipDeviceSynchronize failed: %s\n", hipGetErrorString(err));
        goto cleanup;
    }

    printf("Kernel executed successfully!\n");

    // Copy result back to host
    err = hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("hipMemcpy back to host failed: %s\n", hipGetErrorString(err));
        goto cleanup;
    }

    // Print results
    printf("Vector Addition Results:\n");
    for (int i = 0; i < N; i++)
    {
        printf("%.1f + %.1f = %.1f\n", h_a[i], h_b[i], h_c[i]);
    }

    printf("CUDA vector addition completed successfully!\n");

cleanup:
    if (d_a)
        hipFree(d_a);
    if (d_b)
        hipFree(d_b);
    if (d_c)
        hipFree(d_c);

    return (err == hipSuccess) ? 0 : 1;
}
